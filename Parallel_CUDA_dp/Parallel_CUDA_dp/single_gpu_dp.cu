
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

hipError_t cudaDotProduct(int *c, const int *a, const int *b, unsigned int size);
int* allocAndAssignMat(int size);

__global__ void dot(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] += a[i] * b[i];
}

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	const int N = 10000;  // this is the sqrt of the total elements or the len of one side of the square matrix
	const int* a = allocAndAssignMat(N * N);
	const int* b = allocAndAssignMat(N * N);
	int* c = (int*)malloc((N * N) * sizeof(int));

	for (int i = 0; i < N * N; i++) {
		c[i] = 0;
	}
    
    int mySum = 0;

	hipEventRecord(start);

    // Add vectors in parallel.
    hipError_t cudaStatus = cudaDotProduct(c, a, b, N*N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaDotProduct failed!");
        return 1;
    }
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	for (int i = 0; i < N*N; i++) {
		//printf("%d ", c[i]);
		mySum += c[i];
	}

    //Results
	printf("Size of N*N: %d \nResult: %d \nTime in kernel %f \n", N * N, mySum, milliseconds);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t cudaDotProduct(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));  // allocating the space on the gpu
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);  // moving the data to the gpu counterpart not c as that is results
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    dot<<<(size+255)/256, 256>>>(dev_c, dev_a, dev_b);  // execution configuration - 

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

int* allocAndAssignMat(int size) {
	/*
		This function takes in the size of the matrix (N*N) and returns a pointer with appropriate memory allocated as well as filled with values

		@params: int size
		@returns: int* ptr
	*/
	int* ptr = (int*)malloc(size * sizeof(int));
	for (int i = 0; i < size; i++) {
		ptr[i] = 2;
	}
	return ptr;
}
